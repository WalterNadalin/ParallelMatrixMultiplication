#include <hip/hip_runtime.h>

#include "hipblas.h"

extern "C" void cuda_multiplication(double *A, int m, int n, double *B, int k, double *C, float *io_time, float *cp_time)
{
  /* ... load CPU data into GPU buffers a_gpu and b_gpu */
  double *devA, *devB, *devC; // Matrices on the device
  float time;
  const double alpha = 1, beta = 0; // Parameters for `cublasDgemm`
  hipblasHandle_t handle;
  hipEvent_t start, stop;

  hipMalloc((void**)&devA, n * m * sizeof(double));
  hipMalloc((void**)&devB, n * k * sizeof(double));
  hipMalloc((void**)&devC, m * k * sizeof(double));

  hipblasCreate(&handle);
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0); 
  hipblasSetMatrix(m, n, sizeof(double), A, m, devA, m);
  hipblasSetMatrix(n, k, sizeof(double), B, n, devB, n);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  *io_time += time / 1000;
	
  hipEventRecord(start, 0);
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, k, m, n, &alpha, devB, k, devA, n, &beta, devC, k);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  *cp_time += time / 1000;
  
  hipEventRecord(start, 0);
  hipblasGetMatrix(k, m, sizeof(double), devC, k, C, n);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  *io_time += time / 1000;

  hipFree(devA);
  hipFree(devB);
  hipFree(devC);
  hipblasDestroy(handle);
}
