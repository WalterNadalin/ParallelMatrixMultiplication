#include <hip/hip_runtime.h>

#include "hipblas.h"

extern "C" void cuda_multiplication(double *A, int m, int n, double *B, int k, double *C)
{
  /* ... load CPU data into GPU buffers a_gpu and b_gpu */
  double *devA, *devB, *devC; // Matrices on the device
  hipblasHandle_t handle;
  //cublasStatus_t stat;
  const double alpha = 1, beta = 0; // Parameters for `cublasDgemm`

  hipMalloc((void**)&devA, n * m * sizeof(double));
  hipMalloc((void**)&devB, n * k * sizeof(double));
  hipMalloc((void**)&devC, m * k * sizeof(double));

  hipblasCreate(&handle);

  hipblasSetMatrix(m, n, sizeof(double), A, m, devA, m);
  hipblasSetMatrix(n, k, sizeof(double), B, n, devB, n);

  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, k, m, n, &alpha, devB, k, devA, n, &beta, devC, k);

 // safecall(cudaThreadSynchronize());
 // safecall(cudaGetLastError());
    
  hipblasGetMatrix(k, m, sizeof(double), devC, k, C, n);

  hipFree(devA);
  hipFree(devB);
  hipFree(devC);
  hipblasDestroy(handle);
}
